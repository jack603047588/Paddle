#include "hip/hip_runtime.h"
/* Copyright (c) 2022 PaddlePaddle Authors. All Rights Reserved.

Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at

    http://www.apache.org/licenses/LICENSE-2.0

Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/operators/alias_method_op.h"

#include <hiprand/hiprand_kernel.h>
#include <algorithm>
#include <random>
#include <sstream>
#include <string>

#include "paddle/fluid/platform/device/gpu/gpu_primitives.h"

namespace paddle {
namespace operators {

constexpr int CUDA_NUM_THREADS = platform::PADDLE_CUDA_NUM_THREADS;
#define GET_BLOCK(N) ((N + CUDA_NUM_THREADS - 1) / CUDA_NUM_THREADS)

#define CUDA_KERNEL_LOOP(i, n)                                  \
  for (auto i = blockIdx.x * blockDim.x + threadIdx.x; i < (n); \
       i += blockDim.x * gridDim.x)
#define CUDA_BLOCK(N) GET_BLOCK(N), CUDA_NUM_THREADS, 0

__global__ void SetData(int64_t* tab, const float* idx, uint64_t n) {
  CUDA_KERNEL_LOOP(i, n) { tab[int64_t(idx[i])] = 1; }
}

__global__ void RejectSampling(float* out, const int num, const float* accept,
                               const float* alias, const size_t len,
                               const int device_id, const int device_num,
                               int64_t* noids, uint64_t seed) {
  // https://docs.nvidia.com/cuda/hiprand/device-api-overview.html#distributions

  hiprandStatePhilox4_32_10_t state;
  int seq = blockDim.x * blockIdx.x + threadIdx.x;
  hiprand_init(seed, seq, 0, &state);

  CUDA_KERNEL_LOOP(j, num) {
    while (true) {
      float i = hiprand_uniform(&state);                     // (0, 1]
      int r = std::ceil(hiprand_uniform(&state) * len) - 1;  // [0, n)

      int s = i <= accept[r] ? r : alias[r];
      // printf("i:%f r:%d acc:%f ali:%f s: %d\n",
      //        i, r, accept[r], alias[r], s);
      // if (s % device_num == device_id && noids[s] == 0) {
      if (noids[s] == 0) {
        out[j] = static_cast<float>(s);
        break;
      }
    }
  }
}

template <typename T>
class AliasMethodCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext& context) const override {
    auto place = context.GetPlace();
    int device_id = place.GetDeviceId();
    int device_num = platform::GetGPUDeviceCount();
    auto stream = dynamic_cast<phi::GPUContext*>(
                  platform::DeviceContextPool::Instance().Get(place))
                  ->stream();

    const framework::Tensor* accept =
        context.Input<framework::Tensor>("Accept");
    const framework::Tensor* alias = context.Input<framework::Tensor>("Alias");
    const framework::Tensor* noids = context.Input<framework::Tensor>("Noids");
    framework::Tensor* out = context.Output<framework::Tensor>("Out");
    int num = context.Attr<int>("Num");

    framework::Tensor table;
    int64_t* d_noids =
        table.mutable_data<int64_t>({accept->numel(), 1}, context.GetPlace());
    hipMemsetAsync(d_noids, 0, accept->numel() * sizeof(int64_t), stream);
    SetData<<<CUDA_BLOCK(noids->numel()), stream>>>(
        d_noids, noids->data<float>(), noids->numel());

    uint64_t seed = std::random_device()();
    float* d_out = out->mutable_data<float>(context.GetPlace());
    RejectSampling<<<CUDA_BLOCK(num), stream>>>(
        d_out, num, accept->data<float>(), alias->data<float>(),
        accept->numel(), device_id, device_num, d_noids, seed);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(alias_method, ops::AliasMethodCUDAKernel<float>)
